// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256

//@@ insert code here

int main(int argc, char **argv) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage    = wbImport(inputImageFile);
  imageWidth    = wbImage_getWidth(inputImage);
  imageHeight   = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage   = wbImage_new(imageWidth, imageHeight, imageChannels);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  wbSolution(args, outputImage);

  //@@ insert code here

  return 0;
}
