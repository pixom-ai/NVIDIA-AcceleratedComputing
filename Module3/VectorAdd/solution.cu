#include "hip/hip_runtime.h"
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < len) {
    out[index] = in1[index] + in2[index];
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput1, inputLength * sizeof(float));
  hipMalloc((void **)&deviceInput2, inputLength * sizeof(float));
  hipMalloc((void **)&deviceOutput, inputLength * sizeof(float));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(32);
  dim3 gridDim(ceil(((float)inputLength) / ((float)blockDim.x)));

  wbLog(TRACE, "Block dimension is ", blockDim.x);
  wbLog(TRACE, "Grid dimension is ", gridDim.x);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<gridDim, blockDim>>>(deviceInput1, deviceInput2, deviceOutput,
                                inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
