#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y,
      Row = by * TILE_WIDTH + ty, Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;

  for (int m = 0; m < (numAColumns - 1) / TILE_WIDTH + 1; ++m) {
    if (Row < numARows && m * TILE_WIDTH + tx < numAColumns)
      ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
    else
      ds_M[ty][tx] = 0;
    if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows)
      ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
    else
      ds_N[ty][tx] = 0;

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k)
      Pvalue += ds_M[ty][k] * ds_N[k][tx];
    __syncthreads();
  }
  if (Row < numCRows && Col < numCColumns)
    C[Row * numCColumns + Col] = Pvalue;
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(sizeof(float) * numCRows * numCColumns);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc(&deviceA, sizeof(float) * numARows * numAColumns);
  hipMalloc(&deviceB, sizeof(float) * numBRows * numBColumns);
  hipMalloc(&deviceC, sizeof(float) * numCRows * numCColumns);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns,
             hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns,
             hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid((numCColumns - 1) / TILE_WIDTH + 1,
               (numCRows - 1) / TILE_WIDTH + 1, 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<dimGrid, dimBlock>>>(
      deviceA, deviceB, deviceC, numARows, numAColumns, numBRows,
      numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns,
             hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
