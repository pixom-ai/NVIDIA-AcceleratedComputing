#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define Mask_width 5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

//@@ INSERT CODE HERE
__global__ void convolution(float *I, const float *__restrict__ M,
                            float *P, int channels, int width,
                            int height) {
  __shared__ float N_ds[w][w];
  int k;
  for (k = 0; k < channels; k++) {
    // First batch loading
    int dest = threadIdx.y * TILE_WIDTH + threadIdx.x, destY = dest / w,
        destX = dest % w,
        srcY  = blockIdx.y * TILE_WIDTH + destY - Mask_radius,
        srcX  = blockIdx.x * TILE_WIDTH + destX - Mask_radius,
        src   = (srcY * width + srcX) * channels + k;
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) {
      N_ds[destY][destX] = I[src];
    } else {
      N_ds[destY][destX] = 0;
    }

    // Second batch loading
    dest =
        threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
    destY = dest / w, destX = dest % w;
    srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius;
    srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius;
    src  = (srcY * width + srcX) * channels + k;
    if (destY < w) {
      if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) {
        N_ds[destY][destX] = I[src];
      } else {
        N_ds[destY][destX] = 0;
      }
    }
    __syncthreads();

    float accum = 0;
    int y, x;
    for (y = 0; y < Mask_width; y++) {
      for (x = 0; x < Mask_width; x++) {
        accum +=
            N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * Mask_width + x];
      }
    }
    y = blockIdx.y * TILE_WIDTH + threadIdx.y;
    x = blockIdx.x * TILE_WIDTH + threadIdx.x;
    if (y < height && x < width)
      P[(y * width + x) * channels + k] = clamp(accum);
    __syncthreads();
  }
}

int main(int argc, char *argv[]) {
  wbArg_t arg;
  int maskRows;
  int maskColumns;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  char *inputMaskFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *hostMaskData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *deviceMaskData;

  arg = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(arg, 0);
  inputMaskFile  = wbArg_getInputFile(arg, 1);

  inputImage   = wbImport(inputImageFile);
  hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

  assert(maskRows == 5);    /* mask height is fixed to 5 in this mp */
  assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

  imageWidth    = wbImage_getWidth(inputImage);
  imageHeight   = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceMaskData,
             maskRows * maskColumns * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceMaskData, hostMaskData,
             maskRows * maskColumns * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 dimGrid(ceil((float)imageWidth / TILE_WIDTH),
               ceil((float)imageHeight / TILE_WIDTH));
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData,
                                     deviceOutputImageData, imageChannels,
                                     imageWidth, imageHeight);
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(arg, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);
  hipFree(deviceMaskData);

  free(hostMaskData);
  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
