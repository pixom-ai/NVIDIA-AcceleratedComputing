#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < len)
    out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  hipStream_t stream[4];
  float *d_A[4], *d_B[4], *d_C[4];
  int i, k, Seglen = 1024;
  int Gridlen = (Seglen - 1) / 256 + 1;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc((4*Seglen+inputLength) * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");
  
  
  for (i = 0; i < 4; i++) {
    hipStreamCreate(&stream[i]);
    wbCheck(hipMalloc((void **)&d_A[i], (4*Seglen+inputLength) * sizeof(float)));
    wbCheck(hipMalloc((void **)&d_B[i], (4*Seglen+inputLength) * sizeof(float)));
    wbCheck(hipMalloc((void **)&d_C[i], (4*Seglen+inputLength) * sizeof(float)));
  }

  for (i = 0; i < inputLength; i += Seglen * 4) {
    for (k = 0; k < 4; k++) {
      hipMemcpyAsync(d_A[k], hostInput1 + i + k * Seglen,
                      Seglen * sizeof(float), hipMemcpyHostToDevice,
                      stream[k]);
      hipMemcpyAsync(d_B[k], hostInput2 + i + k * Seglen,
                      Seglen * sizeof(float), hipMemcpyHostToDevice,
                      stream[k]);
      vecAdd<<<Gridlen, 256, 0, stream[k]>>>(d_A[k], d_B[k], d_C[k],
                                             Seglen);
    }
    hipStreamSynchronize(stream[0]);
    hipStreamSynchronize(stream[1]);
    hipStreamSynchronize(stream[2]);
    hipStreamSynchronize(stream[3]);
    for (k = 0; k < 4; k++) {
      hipMemcpyAsync(hostOutput + i + k * Seglen, d_C[k],
                      Seglen * sizeof(float), hipMemcpyDeviceToHost,
                      stream[k]);
    }
  }
  hipDeviceSynchronize();

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  hipStreamDestroy(stream[2]);
  hipStreamDestroy(stream[3]);

  for (k = 0; k < 4; k++) {
    hipFree(d_A[k]);
    hipFree(d_B[k]);
    hipFree(d_C[k]);
  }

  return 0;
}
